#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, int* img, float stepX, float stepY, int maxIterations, int pitch, int height) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + stepX * thisX;
    float y = lowerY + stepY * thisY;

    int *ptr = (int *)((char *) img + thisY * pitch) + thisX;
    *ptr = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);

    int *h_img;
    hipHostAlloc((void **) &h_img, size, hipHostMallocMapped);

    int *d_img;
    size_t pitch;
    hipMallocPitch((void **) &d_img, &pitch, sizeof(int) * resX, resY);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(resX / BLOCK_SIZE, resY / BLOCK_SIZE);
    mandelKernel<<<grid, block>>>(lowerX, lowerY, d_img, stepX, stepY, maxIterations, pitch, resY);

    hipMemcpy2D(h_img, sizeof(int) * resX, d_img, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);
    hipFree(d_img);
    
    memcpy(img, h_img, size);
    hipHostFree(h_img);
}