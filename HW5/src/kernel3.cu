#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 64000

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, int* img, float stepX, float stepY, int resX, int resY, int maxIterations, int group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    int group = thisY * resX + thisX;
    if (group >= group_size) return;

    for (int i = group; i < resX * resY; i+= group_size) {
      thisX = i % resX;
      thisY = i / resX;
      float x = lowerX + stepX * thisX;
      float y = lowerY + stepY * thisY;

      img[i] = mandel(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);

    int *h_img;
    hipHostAlloc((void **) &h_img, size, hipHostMallocMapped);

    int *d_img;
    size_t pitch;
    hipMallocPitch((void **) &d_img, &pitch, resX * resY, sizeof(int));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(resX / BLOCK_SIZE, resY / BLOCK_SIZE);
    mandelKernel<<<grid, block>>>(lowerX, lowerY, d_img, stepX, stepY, resX, resY, maxIterations, GROUP_SIZE);

    hipMemcpy(h_img, d_img, size, hipMemcpyDeviceToHost);
    hipFree(d_img);
    
    memcpy(img, h_img, size);
    hipHostFree(h_img);
}